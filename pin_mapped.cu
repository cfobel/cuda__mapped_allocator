#include "hip/hip_runtime.h"
// incrementMappedArrayInPlace.cu
#include <iostream>
#include <cstdio>
#include <cassert>
#include <boost/format.hpp>
#include <hip/hip_runtime.h>
using namespace std;

#define _ boost::format
 
// define the problem and block size
#define NUMBER_OF_ARRAY_ELEMENTS 100000
#define N_THREADS_PER_BLOCK 256
 
void incrementArrayOnHost(float *a, int N)
{
  int i;
  for (i=0; i < N; i++) a[i] = a[i]+1.f;
}
 
__global__ void incrementArrayOnDevice(float *a, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) a[idx] = a[idx]+1.f;
}
 
void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }                        
}
 

int main(int argc, char** argv) {
	int major = 0;
    int minor = 0;
	int deviceCount = 0;

	hipError_t err = hipInit(0);
    hipGetDeviceCount(&deviceCount);

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
        return 0;
	}

    // This function call returns 9999 for both major & minor fields, 
    // if no CUDA capable devices are present
    hipDeviceComputeCapability(&major, &minor, 0);


    float *a_m; // pointer to host memory
    float *a_d; // pointer to mapped device memory
    float *check_h;   // pointer to host memory used to check results
    int i, N = NUMBER_OF_ARRAY_ELEMENTS;
    size_t size = N*sizeof(float);
    hipDeviceProp_t deviceProp;
    
    #if CUDART_VERSION < 2020
    #error "This CUDART version does not support mapped memory!\n"
    #endif
    
    // Get properties and verify device 0 supports mapped memory
    hipGetDeviceProperties(&deviceProp, 0);
    checkCUDAError("hipGetDeviceProperties");
    
    if(!deviceProp.canMapHostMemory) {
        fprintf(stderr, "Device %d cannot map host memory!\n", 0);
        exit(EXIT_FAILURE);
    }
    
    // set the device flags for mapping host memory
    hipSetDeviceFlags(hipDeviceMapHost);
    checkCUDAError("hipSetDeviceFlags");
    
    // allocate mapped arrays
    cout << "allocate mapped arrays...";
    cout.flush();
    hipHostAlloc((void **)&a_m, size, hipHostMallocMapped);
    checkCUDAError("hipHostMallocMapped");
    cout << "DONE" << endl;
    
    // Get the device pointers to the mapped memory
    cout << "Get the device pointers to the mapped memory...";
    cout.flush();
    hipHostGetDevicePointer((void **)&a_d, (void *)a_m, 0);
    checkCUDAError("hipHostGetDevicePointer");
    cout << "DONE" << endl;
    
    // initialization of host data
    cout << "Initialize host data...";
    cout.flush();
    for (i=0; i<N; i++) a_m[i] = (float)i;
    cout << "DONE" << endl;
    
    // do calculation on device:
    // Part 1 of 2. Compute execution configuration
    int blockSize = N_THREADS_PER_BLOCK;
    int nBlocks = N/blockSize + (N%blockSize > 0?1:0);
    
    // Part 2 of 2. Call incrementArrayOnDevice kernel
    cout << "Request calculation on GPU...";
    cout.flush();
    incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N);
    checkCUDAError("incrementArrayOnDevice");
    cout << "DONE" << endl;
    
    /* Note the allocation, initialization and call to incrementArrayOnHost
        occurs asynchronously to the GPU */
    cout << "Perform calculation on CPU...";
    cout.flush();
    check_h = (float *)malloc(size);
    for (i=0; i<N; i++) check_h[i] = (float)i;
    incrementArrayOnHost(check_h, N);
    cout << "DONE" << endl;
    
    // Make certain that all threads are idle before proceeding
    cout << "Wait for GPU calculation to finish...";
    cout.flush();
    hipDeviceSynchronize();
    checkCUDAError("hipDeviceSynchronize");
    cout << "DONE" << endl;
    
    // check results
    cout << "Verify results...";
    cout.flush();
    for (i=0; i<N; i++) assert(check_h[i] == a_m[i]);
    cout << "DONE" << endl;
    
    // cleanup
    free(check_h); // free host memory
    hipHostFree(a_m); // free mapped memory (and device pointers)
}
